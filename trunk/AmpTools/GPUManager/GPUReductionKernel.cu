#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/*
    Parallel reduction kernels
*/


#ifndef __GPU_REDUCE_KERNEL_H__
#define __GPU_REDUCE_KERNEL_H__


#define SMVERSION sm13 //HHM ADDITION!!!

#include <stdio.h>
#include "GPUSharedMem.cuh"


#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif

// Macros to append an SM version identifier to a function name
// This allows us to compile a file multiple times for different architecture
// versions
// The second macro is necessary to evaluate the value of the SMVERSION macro
// rather than appending "SMVERSION" itself


#define FUNCVERSION(x, y) x ## _ ## y
#define XFUNCVERSION(x, y) FUNCVERSION(x, y)
#define FUNC(NAME) XFUNCVERSION(NAME, SMVERSION) 

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved 
   inactivity means that no whole warps are active, which is also very 
   inefficient */
template <class T, unsigned int blockSize>
__global__ void
FUNC(reduce5)(T *g_idata, T *g_odata)
{
    SharedMemory<T> smem;
    T *sdata = smem.getPointer();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i+blockSize];
    __syncthreads();

    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }
    
#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        if (blockSize >=  64) { sdata[tid] += sdata[tid + 32]; EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] += sdata[tid + 16]; EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] += sdata[tid +  8]; EMUSYNC; }
        if (blockSize >=   8) { sdata[tid] += sdata[tid +  4]; EMUSYNC; }
        if (blockSize >=   4) { sdata[tid] += sdata[tid +  2]; EMUSYNC; }
        if (blockSize >=   2) { sdata[tid] += sdata[tid +  1]; EMUSYNC; }
    }
    
    // write result for this block to global mem 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/*
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)
*/
template <class T, unsigned int blockSize>
__global__ void
FUNC(reduce6)(T *g_idata, T *g_odata, unsigned int n)
{
    SharedMemory<T> smem;
    T *sdata = smem.getPointer();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridSize).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        sdata[tid] += g_idata[i] + g_idata[i+blockSize];  
        i += gridSize;
    } 
    __syncthreads();

    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }
    
#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        if (blockSize >=  64) { sdata[tid] += sdata[tid + 32]; EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] += sdata[tid + 16]; EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] += sdata[tid +  8]; EMUSYNC; }
        if (blockSize >=   8) { sdata[tid] += sdata[tid +  4]; EMUSYNC; }
        if (blockSize >=   4) { sdata[tid] += sdata[tid +  2]; EMUSYNC; }
        if (blockSize >=   2) { sdata[tid] += sdata[tid +  1]; EMUSYNC; }
    }
    
    // write result for this block to global mem 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T>
void 
FUNC(reduce)(int size, int threads, int blocks, 
             int whichKernel, T *d_idata, T *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = threads * sizeof(T);

    // choose which of the optimized versions of reduction to launch
    switch (whichKernel)
    {
        case 5:
        switch (threads)
        {
        case 512:
            FUNC(reduce5)<T, 512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 256:
            FUNC(reduce5)<T, 256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 128:
            FUNC(reduce5)<T, 128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 64:
            FUNC(reduce5)<T,  64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 32:
            FUNC(reduce5)<T,  32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 16:
            FUNC(reduce5)<T,  16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  8:
            FUNC(reduce5)<T,   8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  4:
            FUNC(reduce5)<T,   4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  2:
            FUNC(reduce5)<T,   2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  1:
            FUNC(reduce5)<T,   1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        }
        break;       
    case 6:
    default:
        switch (threads)
        {
        case 512:
            FUNC(reduce6)<T, 512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 256:
            FUNC(reduce6)<T, 256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 128:
            FUNC(reduce6)<T, 128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 64:
            FUNC(reduce6)<T,  64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 32:
            FUNC(reduce6)<T,  32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 16:
            FUNC(reduce6)<T,  16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  8:
            FUNC(reduce6)<T,   8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  4:
            FUNC(reduce6)<T,   4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  2:
            FUNC(reduce6)<T,   2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  1:
            FUNC(reduce6)<T,   1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        }
        break;       
    }
}

extern "C"
void FUNC(reduceInt)(int size, int threads, int blocks, 
                     int whichKernel, int *d_idata, int *d_odata)
{
    FUNC(reduce)<int>(size, threads, blocks, whichKernel, d_idata, d_odata);
}

extern "C"
void FUNC(reduceFloat)(int size, int threads, int blocks, 
                       int whichKernel, float *d_idata, float *d_odata)
{
    FUNC(reduce)<float>(size, threads, blocks, whichKernel, d_idata, d_odata);
}

extern "C"
void FUNC(reduceDouble)(int size, int threads, int blocks, 
                        int whichKernel, double *d_idata, double *d_odata)
{
    FUNC(reduce)<double>(size, threads, blocks, whichKernel, d_idata, d_odata);
}

#endif // __GPU_REDUCE_KERNEL_H__
