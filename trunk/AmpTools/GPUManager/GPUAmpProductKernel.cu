#include "hip/hip_runtime.h"
//******************************************************************************
// This file is part of AmpTools, a package for performing Amplitude Analysis
// 
// Copyright Trustees of Indiana University 2010, all rights reserved
// 
// This software written by Matthew Shepherd, Ryan Mitchell, and 
//                  Hrayr Matevosyan at Indiana University, Bloomington
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
// 1. Redistributions of source code must retain the above copyright
//    notice and author attribution, this list of conditions and the
//    following disclaimer. 
// 2. Redistributions in binary form must reproduce the above copyright
//    notice and author attribution, this list of conditions and the
//    following disclaimer in the documentation and/or other materials
//    provided with the distribution.
// 3. Neither the name of the University nor the names of its contributors
//    may be used to endorse or promote products derived from this software
//    without specific prior written permission.
// 
// Creation of derivative forms of this software for commercial
// utilization may be subject to restriction; written permission may be
// obtained from the Trustees of Indiana University.
// 
// INDIANA UNIVERSITY AND THE AUTHORS MAKE NO REPRESENTATIONS OR WARRANTIES, 
// EXPRESS OR IMPLIED.  By way of example, but not limitation, INDIANA 
// UNIVERSITY MAKES NO REPRESENTATIONS OR WARRANTIES OF MERCANTABILITY OR 
// FITNESS FOR ANY PARTICULAR PURPOSE OR THAT THE USE OF THIS SOFTWARE OR 
// DOCUMENTATION WILL NOT INFRINGE ANY PATENTS, COPYRIGHTS, TRADEMARKS, 
// OR OTHER RIGHTS.  Neither Indiana University nor the authors shall be 
// held liable for any liability with respect to any claim by the user or 
// any other party arising from use of the program.
//******************************************************************************

#include "GPUCustomTypes.h"

__constant__ int da_iNAmps;  
__constant__ int da_iNEvents; // number of events padded to the closest 2^n
__constant__ GDouble da_pfDevVRe[GPU_MAX_AMPS]; // fit parameters stored in
__constant__ GDouble da_pfDevVIm[GPU_MAX_AMPS]; // const memory space

extern "C" GDouble* da_pfDevVRe_addr() { return da_pfDevVRe;  }
extern "C" GDouble* da_pfDevVIm_addr() { return da_pfDevVIm;  }
extern "C" int*     da_iNAmps_addr()   { return &da_iNAmps;   }
extern "C" int*     da_iNEvents_addr() { return &da_iNEvents; }



__global__ void
amp_kernel( GDouble* pfDevAmps, GDouble* pfDevWeights, GDouble* pfDevRes )
{
	int i = threadIdx.x + GPU_BLOCK_SIZE_X * threadIdx.y + 
            ( blockIdx.x + blockIdx.y * gridDim.x ) * GPU_BLOCK_SIZE_SQ;

	int iA, iB;
	GDouble fSumRe = 0.;

  // index to amplitude A_alpha for the ith event
  int aIndA = 2*i;

	for( iA = 0; iA < da_iNAmps; ++iA ){

  // index to amplitude A_alpha for the ith event
  // (reduce computations by incrementing at end of loop)
  //	  int aIndA = i + 2*da_iNEvents*iA;

    // index in the array of V_alpha * conj( V_beta )
    int vInd = iA*(iA+1)/2;
    
    // index to amplitude A_beta for the ith event
    int aIndB = 2*i;
	  for( iB = 0; iB <= iA; ++iB ){

      // index in the array of V_alpha * conj( V_beta )
      // (reduce computations by incrementing at end of loop)
      //	   int vInd = iA*(iA+1)/2+iB;

	    // index to amplitude A_beta for the ith event
      // (reduce computations by incrementing at end of loop)
      //     int aIndB = i + 2*da_iNEvents*iB;

      // only compute the real part of the intensity since
      // the imaginary part should sum to zero
	    GDouble term = da_pfDevVRe[vInd] * 
               ( pfDevAmps[aIndA]   * pfDevAmps[aIndB] +
                 pfDevAmps[aIndA+1] * pfDevAmps[aIndB+1] );

      term -= da_pfDevVIm[vInd] *
               ( pfDevAmps[aIndA+1] * pfDevAmps[aIndB] -
                 pfDevAmps[aIndA]   * pfDevAmps[aIndB+1] );

	    // we're only summing over the lower diagonal so we need
      // to double the contribution for off diagonal elements
      if( iA != iB ) term *= 2;

 	    fSumRe += term;

      ++vInd;
      aIndB += 2*da_iNEvents;
	  }
    
    aIndA += 2*da_iNEvents;
	}
	
	pfDevRes[i] = pfDevWeights[i] * G_LOG( fSumRe );
}

extern "C" void GPU_ExecAmpKernel( dim3 dimGrid, dim3 dimBlock, 
     GDouble* pfDevAmps, GDouble* pfDevWeights, GDouble* pfDevRes )
{
	amp_kernel<<< dimGrid, dimBlock >>>( pfDevAmps, pfDevWeights, pfDevRes );
}




__global__ void
int_element_kernel( int iA, int iB, GDouble* pfDevAmps,
                    GDouble* pfDevWeights, GDouble* pfDevResRe,
                    GDouble* pfDevResIm )
{
	int i = threadIdx.x + GPU_BLOCK_SIZE_X * threadIdx.y + 
            ( blockIdx.x + blockIdx.y * gridDim.x ) * GPU_BLOCK_SIZE_SQ;

  int aInd = 2*i + 2*da_iNEvents*iA;
  int bInd = 2*i + 2*da_iNEvents*iB;

  pfDevResRe[i] = pfDevWeights[i] * (
                    pfDevAmps[aInd]   * pfDevAmps[bInd]  +
                    pfDevAmps[aInd+1] * pfDevAmps[bInd+1] );
  
  pfDevResIm[i] = pfDevWeights[i] * (
                    pfDevAmps[aInd+1] * pfDevAmps[bInd] -
                    pfDevAmps[aInd]   * pfDevAmps[bInd+1] );
}

extern "C" void GPU_ExecIntElementKernel( dim3 dimGrid, dim3 dimBlock,
     int iA, int iB, GDouble* pfDevAmps, GDouble* pfDevWeights, 
     GDouble* pfDevResRe, GDouble* pfDevResIm )
{
	int_element_kernel<<< dimGrid, dimBlock >>>( iA, iB, pfDevAmps, pfDevWeights,
                                               pfDevResRe, pfDevResIm );
}

